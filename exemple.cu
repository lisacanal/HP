#include "hip/hip_runtime.h"
// #define STB_IMAGE_IMPLEMENTATION
// #define STB_IMAGE_WRITE_IMPLEMENTATION

// #include "stb_image.h"
// #include "stb_image_write.h"
// #include <stdlib.h>
// #include <math.h>
// #include <stdio.h>

// // Gaussian function
// double gaussian(double x, double sigma) {
//     return exp(-(x * x) / (2.0 * sigma * sigma));
// }

// // Function to precompute spatial Gaussian weights
// void compute_spatial_weights(double *spatial_weights, int d, double sigma_space) {
//     int radius = d / 2;
//     for (int i = 0; i < d; i++) {
//         for (int j = 0; j < d; j++) {
//             int x = i - radius, y = j - radius;
//             spatial_weights[i * d + j] = gaussian(sqrt(x * x + y * y), sigma_space);
//         }
//     }
// }

// // Manual bilateral filter
// void bilateral_filter(unsigned char *src, unsigned char *dst, int width, int height, int channels, int d, double sigma_color, double sigma_space) {
//     int radius = d / 2;

//     // Precompute spatial Gaussian weights
//     double *spatial_weights = (double *)malloc(d * d * sizeof(double));
//     if (!spatial_weights) {
//         printf("Memory allocation for spatial weights failed!\n");
//         return;
//     }
//     compute_spatial_weights(spatial_weights, d, sigma_space);

//     // Process image
//     for (int y = radius; y < height - radius; y++) {
//         for (int x = radius; x < width - radius; x++) {
//             double weight_sum[3] = {0.0, 0.0, 0.0};
//             double filtered_value[3] = {0.0, 0.0, 0.0};

//             // Get center pixel pointer
//             unsigned char *center_pixel = src + (y * width + x) * channels;

//             // Iterate over local window
//             for (int i = 0; i < d; i++) {
//                 for (int j = 0; j < d; j++) {
//                     int nx = x + j - radius;
//                     int ny = y + i - radius;

//                     // Bounds check to ensure we're within the image
//                     if (nx < 0 || nx >= width || ny < 0 || ny >= height) {
//                         continue;
//                     }

//                     // Get neighbor pixel pointer
//                     unsigned char *neighbor_pixel = src + (ny * width + nx) * channels;

//                     for (int c = 0; c < channels; c++) {
//                         // Compute range weight
//                         double range_weight = gaussian(abs(neighbor_pixel[c] - center_pixel[c]), sigma_color);
//                         double weight = spatial_weights[i * d + j] * range_weight;

//                         // Accumulate weighted sum
//                         filtered_value[c] += neighbor_pixel[c] * weight;
//                         weight_sum[c] += weight;
//                     }
//                 }
//             }

//             // Normalize and store result
//             unsigned char *output_pixel = dst + (y * width + x) * channels;
//             for (int c = 0; c < channels; c++) {
//                 output_pixel[c] = (unsigned char)(filtered_value[c] / (weight_sum[c] + 1e-6)); // Avoid division by zero
//             }
//         }
//     }

//     free(spatial_weights);
// }

// // Main function
// int main(int argc, char *argv[]) {
//     if (argc < 3) {
//         printf("Usage: %s <input_image> <output_image>\n", argv[0]);
//         return 1;
//     }

//     int width, height, channels;
//     unsigned char *image = stbi_load(argv[1], &width, &height, &channels, 0);
//     if (!image) {
//         printf("Error loading image!\n");
//         return 1;
//     }

//     // Ensure that image is not too small for bilateral filter (at least radius of d/2 around edges)
//     if (width <= 5 || height <= 5) {
//         printf("Image is too small for bilateral filter (at least 5x5 size needed).\n");
//         stbi_image_free(image);
//         return 1;
//     }

//     // Allocate memory for output image
//     unsigned char *filtered_image = (unsigned char *)malloc(width * height * channels);
//     if (!filtered_image) {
//         printf("Memory allocation for filtered image failed!\n");
//         stbi_image_free(image);
//         return 1;
//     }
    
//     // Apply the bilateral filter
//     bilateral_filter(image, filtered_image, width, height, channels, 5, 75.0, 75.0);

//     // Save the output image
//     if (!stbi_write_png(argv[2], width, height, channels, filtered_image, width * channels)) {
//         printf("Error saving the image!\n");
//         free(filtered_image);
//         stbi_image_free(image);
//         return 1;
//     }

//     // Free memory
//     stbi_image_free(image);
//     free(filtered_image);

//     printf("Bilateral filtering complete. Output saved as %s\n", argv[2]);
//     return 0;
// }

// #define STB_IMAGE_IMPLEMENTATION
// #define STB_IMAGE_WRITE_IMPLEMENTATION

// #include "stb_image.h"
// #include "stb_image_write.h"
// #include <stdlib.h>
// #include <math.h>
// #include <stdio.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_math_constants.h>

// // Gaussian function
// __device__ double gaussian(double x, double sigma) {
//     return exp(-(x * x) / (2.0 * sigma * sigma));
// }

// // CUDA kernel to compute spatial weights
// __global__ void compute_spatial_weights_kernel(double *spatial_weights, int d, double sigma_space, int radius) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;
    
//     if (i < d && j < d) {
//         int x = i - radius, y = j - radius;
//         spatial_weights[i * d + j] = gaussian(sqrt(x * x + y * y), sigma_space);
//     }
// }

// // CUDA kernel to apply the bilateral filter
// __global__ void bilateral_filter_kernel(unsigned char *src, unsigned char *dst, int width, int height, int channels, int d, double sigma_color, double sigma_space, double *spatial_weights) {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int radius = d / 2;

//     if (x < radius || x >= width - radius || y < radius || y >= height - radius) {
//         return;  // Ignore borders
//     }

//     double weight_sum[3] = {0.0, 0.0, 0.0};
//     double filtered_value[3] = {0.0, 0.0, 0.0};

//     unsigned char *center_pixel = src + (y * width + x) * channels;

//     for (int i = -radius; i <= radius; i++) {
//         for (int j = -radius; j <= radius; j++) {
//             int nx = x + j;
//             int ny = y + i;
            
//             if (nx < 0 || nx >= width || ny < 0 || ny >= height) {
//                 continue; // Skip out-of-bounds pixels
//             }

//             unsigned char *neighbor_pixel = src + (ny * width + nx) * channels;

//             for (int c = 0; c < channels; c++) {
//                 double range_weight = gaussian(abs(neighbor_pixel[c] - center_pixel[c]), sigma_color);
//                 double weight = spatial_weights[(i + radius) * d + (j + radius)] * range_weight;
//                 filtered_value[c] += neighbor_pixel[c] * weight;
//                 weight_sum[c] += weight;
//             }
//         }
//     }

//     unsigned char *output_pixel = dst + (y * width + x) * channels;
//     for (int c = 0; c < channels; c++) {
//         output_pixel[c] = (unsigned char)(filtered_value[c] / (weight_sum[c] + 1e-6));
//     }
// }

// // Function to apply the bilateral filter using CUDA
// void bilateral_filter_cuda(unsigned char *src, unsigned char *dst, int width, int height, int channels, int d, double sigma_color, double sigma_space) {
//     int img_size = width * height * channels;
//     unsigned char *d_src, *d_dst;
//     double *d_spatial_weights;
    
//     hipMalloc((void**)&d_src, img_size);
//     hipMalloc((void**)&d_dst, img_size);
//     hipMalloc((void**)&d_spatial_weights, d * d * sizeof(double));
//     hipMemcpy(d_src, src, img_size, hipMemcpyHostToDevice);
    
//     dim3 threadsPerBlock(16, 16);
//     dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, 
//                    (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

//     compute_spatial_weights_kernel<<<numBlocks, threadsPerBlock>>>(d_spatial_weights, d, sigma_space, d / 2);
//     bilateral_filter_kernel<<<numBlocks, threadsPerBlock>>>(d_src, d_dst, width, height, channels, d, sigma_color, sigma_space, d_spatial_weights);
//     hipMemcpy(dst, d_dst, img_size, hipMemcpyDeviceToHost);
    
//     hipFree(d_src);
//     hipFree(d_dst);
//     hipFree(d_spatial_weights);
// }

// // Main function
// int main(int argc, char *argv[]) {
//     if (argc < 3) {
//         printf("Usage: %s <input_image> <output_image>\n", argv[0]);
//         return 1;
//     }

//     int width, height, channels;
//     unsigned char *image = stbi_load(argv[1], &width, &height, &channels, 0);
//     if (!image) {
//         printf("Error loading image!\n");
//         return 1;
//     }

//     if (width <= 5 || height <= 5) {
//         printf("Image is too small for bilateral filter (at least 5x5 size needed).\n");
//         stbi_image_free(image);
//         return 1;
//     }

//     unsigned char *filtered_image = (unsigned char *)malloc(width * height * channels);
//     if (!filtered_image) {
//         printf("Memory allocation for filtered image failed!\n");
//         stbi_image_free(image);
//         return 1;
//     }
    
//     bilateral_filter_cuda(image, filtered_image, width, height, channels, 5, 75.0, 75.0);

//     if (!stbi_write_png(argv[2], width, height, channels, filtered_image, width * channels)) {
//         printf("Error saving the image!\n");
//         free(filtered_image);
//         stbi_image_free(image);
//         return 1;
//     }

//     stbi_image_free(image);
//     free(filtered_image);

//     printf("Bilateral filtering complete using CUDA. Output saved as %s\n", argv[2]);
//     return 0;
// }

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// Fonction gaussienne
__device__ float calcul_gaussien(float distance, float ecart_type) {
    return expf(-(distance * distance) / (2.0f * ecart_type * ecart_type));
}

// Filtre bilatéral en CUDA
__global__ void filtre_bilateral_cuda(unsigned char *entree, unsigned char *sortie, int largeur, int hauteur, int canaux, int taille_fenetre, float sigma_couleur, float sigma_espace) {
    int coord_x = blockIdx.x * blockDim.x + threadIdx.x;
    int coord_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (coord_x >= largeur || coord_y >= hauteur) return;

    int rayon = taille_fenetre / 2;

    float somme_valeurs[3] = {0.0f, 0.0f, 0.0f};
    float somme_poids[3] = {0.0f, 0.0f, 0.0f};

    unsigned char *pixel_central = entree + (coord_y * largeur + coord_x) * canaux;

    for (int dy = -rayon; dy <= rayon; dy++) {
        for (int dx = -rayon; dx <= rayon; dx++) {
            int voisin_x = coord_x + dx;
            int voisin_y = coord_y + dy;

            if (voisin_x >= 0 && voisin_x < largeur && voisin_y >= 0 && voisin_y < hauteur) {
                unsigned char *pixel_voisin = entree + (voisin_y * largeur + voisin_x) * canaux;

                for (int c = 0; c < canaux; c++) {
                    float poids_spatial = calcul_gaussien(sqrtf((float)(dx * dx + dy * dy)), sigma_espace);
                    float poids_couleur = calcul_gaussien(fabsf((float)pixel_voisin[c] - (float)pixel_central[c]), sigma_couleur);
                    float poids_total = poids_spatial * poids_couleur;

                    somme_valeurs[c] += pixel_voisin[c] * poids_total;
                    somme_poids[c] += poids_total;
                }
            }
        }
    }

    unsigned char *pixel_sortie = sortie + (coord_y * largeur + coord_x) * canaux;
    for (int c = 0; c < canaux; c++) {
        pixel_sortie[c] = (unsigned char)(somme_valeurs[c] / (somme_poids[c] + 1e-6f));
    }
}

// Fonction principale
int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s <image_entree> <image_sortie>\n", argv[0]);
        return 1;
    }

    int largeur, hauteur, canaux;
    unsigned char *image_entree = stbi_load(argv[1], &largeur, &hauteur, &canaux, 0);
    if (!image_entree) {
        printf("Erreur lors du chargement de l’image !\n");
        return 1;
    }

    unsigned char *image_filtrée = (unsigned char *)malloc(largeur * hauteur * canaux);

    unsigned char *gpu_entree, *gpu_sortie;
    hipMalloc(&gpu_entree, largeur * hauteur * canaux);
    hipMalloc(&gpu_sortie, largeur * hauteur * canaux);

    hipMemcpy(gpu_entree, image_entree, largeur * hauteur * canaux, hipMemcpyHostToDevice);

    dim3 taille_bloc(16, 16);
    dim3 taille_grille(32, 32);
    //dim3 taille_grille((largeur + taille_bloc.x - 1) / taille_bloc.x, (hauteur + taille_bloc.y - 1) / taille_bloc.y);

    filtre_bilateral_cuda<<<taille_grille, taille_bloc>>>(gpu_entree, gpu_sortie, largeur, hauteur, canaux, 5, 15.0f, 5.0f);
    hipDeviceSynchronize();
}