#include "hip/hip_runtime.h"
// #define STB_IMAGE_IMPLEMENTATION
// #define STB_IMAGE_WRITE_IMPLEMENTATION

// #include "stb_image.h"
// #include "stb_image_write.h"
// #include <stdlib.h>
// #include <math.h>
// #include <stdio.h>

// // Gaussian function
// double gaussian(double x, double sigma) {
//     return exp(-(x * x) / (2.0 * sigma * sigma));
// }

// // Function to precompute spatial Gaussian weights
// void compute_spatial_weights(double *spatial_weights, int d, double sigma_space) {
//     int radius = d / 2;
//     for (int i = 0; i < d; i++) {
//         for (int j = 0; j < d; j++) {
//             int x = i - radius, y = j - radius;
//             spatial_weights[i * d + j] = gaussian(sqrt(x * x + y * y), sigma_space);
//         }
//     }
// }

// // Manual bilateral filter
// void bilateral_filter(unsigned char *src, unsigned char *dst, int width, int height, int channels, int d, double sigma_color, double sigma_space) {
//     int radius = d / 2;

//     // Precompute spatial Gaussian weights
//     double *spatial_weights = (double *)malloc(d * d * sizeof(double));
//     if (!spatial_weights) {
//         printf("Memory allocation for spatial weights failed!\n");
//         return;
//     }
//     compute_spatial_weights(spatial_weights, d, sigma_space);

//     // Process image
//     for (int y = radius; y < height - radius; y++) {
//         for (int x = radius; x < width - radius; x++) {
//             double weight_sum[3] = {0.0, 0.0, 0.0};
//             double filtered_value[3] = {0.0, 0.0, 0.0};

//             // Get center pixel pointer
//             unsigned char *center_pixel = src + (y * width + x) * channels;

//             // Iterate over local window
//             for (int i = 0; i < d; i++) {
//                 for (int j = 0; j < d; j++) {
//                     int nx = x + j - radius;
//                     int ny = y + i - radius;

//                     // Bounds check to ensure we're within the image
//                     if (nx < 0 || nx >= width || ny < 0 || ny >= height) {
//                         continue;
//                     }

//                     // Get neighbor pixel pointer
//                     unsigned char *neighbor_pixel = src + (ny * width + nx) * channels;

//                     for (int c = 0; c < channels; c++) {
//                         // Compute range weight
//                         double range_weight = gaussian(abs(neighbor_pixel[c] - center_pixel[c]), sigma_color);
//                         double weight = spatial_weights[i * d + j] * range_weight;

//                         // Accumulate weighted sum
//                         filtered_value[c] += neighbor_pixel[c] * weight;
//                         weight_sum[c] += weight;
//                     }
//                 }
//             }

//             // Normalize and store result
//             unsigned char *output_pixel = dst + (y * width + x) * channels;
//             for (int c = 0; c < channels; c++) {
//                 output_pixel[c] = (unsigned char)(filtered_value[c] / (weight_sum[c] + 1e-6)); // Avoid division by zero
//             }
//         }
//     }

//     free(spatial_weights);
// }

// // Main function
// int main(int argc, char *argv[]) {
//     if (argc < 3) {
//         printf("Usage: %s <input_image> <output_image>\n", argv[0]);
//         return 1;
//     }

//     int width, height, channels;
//     unsigned char *image = stbi_load(argv[1], &width, &height, &channels, 0);
//     if (!image) {
//         printf("Error loading image!\n");
//         return 1;
//     }

//     // Ensure that image is not too small for bilateral filter (at least radius of d/2 around edges)
//     if (width <= 5 || height <= 5) {
//         printf("Image is too small for bilateral filter (at least 5x5 size needed).\n");
//         stbi_image_free(image);
//         return 1;
//     }

//     // Allocate memory for output image
//     unsigned char *filtered_image = (unsigned char *)malloc(width * height * channels);
//     if (!filtered_image) {
//         printf("Memory allocation for filtered image failed!\n");
//         stbi_image_free(image);
//         return 1;
//     }
    
//     // Apply the bilateral filter
//     bilateral_filter(image, filtered_image, width, height, channels, 5, 75.0, 75.0);

//     // Save the output image
//     if (!stbi_write_png(argv[2], width, height, channels, filtered_image, width * channels)) {
//         printf("Error saving the image!\n");
//         free(filtered_image);
//         stbi_image_free(image);
//         return 1;
//     }

//     // Free memory
//     stbi_image_free(image);
//     free(filtered_image);

//     printf("Bilateral filtering complete. Output saved as %s\n", argv[2]);
//     return 0;
// }

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// Gaussian function
__device__ double gaussian(double x, double sigma) {
    return exp(-(x * x) / (2.0 * sigma * sigma));
}

// CUDA kernel to compute spatial weights
__global__ void compute_spatial_weights_kernel(double *spatial_weights, int d, double sigma_space, int radius) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < d && j < d) {
        int x = i - radius, y = j - radius;
        spatial_weights[i * d + j] = gaussian(sqrt(x * x + y * y), sigma_space);
    }
}

// CUDA kernel to apply the bilateral filter
__global__ void bilateral_filter_kernel(unsigned char *src, unsigned char *dst, int width, int height, int channels, int d, double sigma_color, double sigma_space, double *spatial_weights) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int radius = d / 2;

    if (x < radius || x >= width - radius || y < radius || y >= height - radius) {
        return;  // Ignore borders
    }

    double weight_sum[3] = {0.0, 0.0, 0.0};
    double filtered_value[3] = {0.0, 0.0, 0.0};

    unsigned char *center_pixel = src + (y * width + x) * channels;

    for (int i = -radius; i <= radius; i++) {
        for (int j = -radius; j <= radius; j++) {
            int nx = x + j;
            int ny = y + i;
            
            if (nx < 0 || nx >= width || ny < 0 || ny >= height) {
                continue; // Skip out-of-bounds pixels
            }

            unsigned char *neighbor_pixel = src + (ny * width + nx) * channels;

            for (int c = 0; c < channels; c++) {
                double range_weight = gaussian(abs(neighbor_pixel[c] - center_pixel[c]), sigma_color);
                double weight = spatial_weights[(i + radius) * d + (j + radius)] * range_weight;
                filtered_value[c] += neighbor_pixel[c] * weight;
                weight_sum[c] += weight;
            }
        }
    }

    unsigned char *output_pixel = dst + (y * width + x) * channels;
    for (int c = 0; c < channels; c++) {
        output_pixel[c] = (unsigned char)(filtered_value[c] / (weight_sum[c] + 1e-6));
    }
}

// Function to apply the bilateral filter using CUDA
void bilateral_filter_cuda(unsigned char *src, unsigned char *dst, int width, int height, int channels, int d, double sigma_color, double sigma_space) {
    int img_size = width * height * channels;
    unsigned char *d_src, *d_dst;
    double *d_spatial_weights;
    
    hipMalloc((void**)&d_src, img_size);
    hipMalloc((void**)&d_dst, img_size);
    hipMalloc((void**)&d_spatial_weights, d * d * sizeof(double));
    hipMemcpy(d_src, src, img_size, hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    compute_spatial_weights_kernel<<<numBlocks, threadsPerBlock>>>(d_spatial_weights, d, sigma_space, d / 2);
    bilateral_filter_kernel<<<numBlocks, threadsPerBlock>>>(d_src, d_dst, width, height, channels, d, sigma_color, sigma_space, d_spatial_weights);
    hipMemcpy(dst, d_dst, img_size, hipMemcpyDeviceToHost);
    
    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_spatial_weights);
}

// Main function
int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s <input_image> <output_image>\n", argv[0]);
        return 1;
    }

    int width, height, channels;
    unsigned char *image = stbi_load(argv[1], &width, &height, &channels, 0);
    if (!image) {
        printf("Error loading image!\n");
        return 1;
    }

    if (width <= 5 || height <= 5) {
        printf("Image is too small for bilateral filter (at least 5x5 size needed).\n");
        stbi_image_free(image);
        return 1;
    }

    unsigned char *filtered_image = (unsigned char *)malloc(width * height * channels);
    if (!filtered_image) {
        printf("Memory allocation for filtered image failed!\n");
        stbi_image_free(image);
        return 1;
    }
    
    bilateral_filter_cuda(image, filtered_image, width, height, channels, 5, 75.0, 75.0);

    if (!stbi_write_png(argv[2], width, height, channels, filtered_image, width * channels)) {
        printf("Error saving the image!\n");
        free(filtered_image);
        stbi_image_free(image);
        return 1;
    }

    stbi_image_free(image);
    free(filtered_image);

    printf("Bilateral filtering complete using CUDA. Output saved as %s\n", argv[2]);
    return 0;
}
